#include <cuda_device_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;




extern "C"{
    void cudaCheckError(hipError_t err,const char* msg){
        if(err != hipSuccess){
            cerr << msg << ": " << hipGetErrorString(err) << endl;
            exit(EXIT_FAILURE);
        }
    }

    __global__ void addKernel(const double *x,const double *y,double *z){
        *z = *x + *y;
    }

    __global__ void subKernel(const double *x,const double *y,double *z){
        *z = *x - *y;
    }

    __global__ void mulKernel(const double *x,const double *y,double *z){
        *z = *x * *y;
    }

    __global__ void tdivKernel(const double *x,const double *y,double *z){
        *z = *x / *y;
    }

    __global__ void fdivKernel(const double *x,const double *y,double *z){
        *z = floor(*x / *y);
    }

    __global__ void powKernel(const double *x,const double *y,double *z){
        *z = pow(*x,*y);
    }

    __global__ void modKernel(const double *x,const double *y,double* z){
        *z = fmod(*x,*y);
    }

    __global__ void eqKernel(const double *x,const double *y,int *z){
        *z = (*x == *y) ? 1 : 0;
    }

    __global__ void neKernel(const double *x,const double *y,int *z){
        *z = (*x != *y) ? 1 : 0;
    }

    __global__ void gtKernel(const double *x,const double *y,int *z){
        *z = (*x > *y) ? 1 : 0;
    }

    __global__ void geKernel(const double *x,const double *y,int *z){
        *z = (*x >= *y) ? 1 : 0;
    }

    __global__ void ltKernel(const double *x,const double *y,int *z){
        *z = (*x < *y) ? 1 : 0;
    }

    __global__ void leKernel(const double *x,const double *y,int *z){
        *z = (*x <= *y) ? 1 : 0;
    }

    __global__ void absKernel(const double *x,double *z){
        if(*x >= 0){
            *z = *x;
        }else{
            *z = -*x;
        }
    }

    __global__ void sqrtKernel(const double *x,double *z){
        *z = __dsqrt_rn(*x);
    }

    __global__ void cbrtKernel(const double *x,double *z){
        *z = cbrt(*x);
    }

    __global__ void factKernel(double *x, double *z) {
        unsigned int thread_id = threadIdx.x;
        unsigned int block_dim = blockDim.x;

        __shared__ double cache[1024];

        double result = 1.0;
        for (int i = thread_id + 1;i <= *x;i += block_dim) {
            result *= i;
        }

        cache[thread_id] = result;
        __syncthreads();

        for (unsigned int i = 1;i < block_dim;i *= 2) {
            if (thread_id % (2 * i) == 0) {
                cache[thread_id] *= cache[thread_id + i];
            }
            __syncthreads();
        }

        if (thread_id == 0) {
            *z = cache[0];
        }
    }

    __global__ void negKernel(const double *x,double* z){
        *z = -*x;
    }

    __global__ void posKernel(const double *x,double* z){
        *z = +*x;
    }




    class Var{
        private:
            double *value;

        public:
            Var(const double data){
                cudaCheckError(hipMalloc(&value,sizeof(double)),"failed to allocate the memory on GPU");                                               // allocate the memory on GPU for double dtype
                cudaCheckError(hipMemcpy(value,&data,sizeof(double),hipMemcpyHostToDevice),"failed to copy data from host to dedvice");               // copy memory from CPU to GPU
            }

            ~Var(){
                cudaCheckError(hipFree(value),"failed to free device memory");                                                                         // delete that memory from the GPU
            }

            double get_value() const {
                double host_value;
                cudaCheckError(hipMemcpy(&host_value,value,sizeof(double),hipMemcpyDeviceToHost),"failed to copy data from device to host");           // copy memory from CPU to GPU
                return host_value;
            }

            Var add(const Var &other) const{
                Var result(0.0);
                addKernel<<<1,1>>>(this -> value,other.value,result.value);
                cudaCheckError(hipGetLastError(),"faield to lauch addKernel");
                cudaCheckError(hipDeviceSynchronize(),"kernel excution failed");
                return result;
            }

            Var sub(const Var &other) const {
                Var result(0.0);
                subKernel<<<1,1>>>(this -> value,other.value,result.value);
                cudaCheckError(hipGetLastError(),"failed to launch subKernel");
                cudaCheckError(hipDeviceSynchronize(),"kernel execution failed");
                return result;
            }

            Var mul(const Var &other) const {
                Var result(0.0);
                mulKernel<<<1,1>>>(this -> value,other.value,result.value);
                cudaCheckError(hipGetLastError(),"failed to launch mulKernel");
                cudaCheckError(hipDeviceSynchronize(),"kernel execution failed");
                return result;
            }

            Var tdiv(const Var &other) const {
                Var result(0.0);
                tdivKernel<<<1,1>>>(this -> value,other.value,result.value);
                cudaCheckError(hipGetLastError(),"failed to laumch tdivKernel");
                cudaCheckError(hipDeviceSynchronize(),"kernel execution failed");
                return result;
            }

            Var fdiv(const Var &other) const {
                Var result(0.0);
                fdivKernel<<<1,1>>>(this -> value,other.value,result.value);
                cudaCheckError(hipGetLastError(),"failed to lauch fdivKernel");
                cudaCheckError(hipDeviceSynchronize(),"kernel execution failed");
                return result;
            }

            Var pow(const Var &other) const {
                Var result(0.0);
                powKernel<<<1,1>>>(this -> value,other.value,result.value);
                cudaCheckError(hipGetLastError(),"failed to launch powKernel");
                cudaCheckError(hipDeviceSynchronize(),"kernel execution failed");
                return result;

            }

            Var mod(const Var &other) const {
                Var result(0.0);
                modKernel<<<1,1>>>(this -> value,other.value,result.value);
                cudaCheckError(hipGetLastError(),"failed to launch modKernel");
                cudaCheckError(hipDeviceSynchronize(),"kernel execution failed");
                return result;
            }

            int eq(const Var &other) const {
                int host_result;
                int *device_result;
                cudaCheckError(hipMalloc(&device_result,sizeof(int)),"failed to allocate the memory on GPU");
                eqKernel<<<1,1>>>(this -> value,other.value,device_result);
                cudaCheckError(hipGetLastError(),"failed to launch eqKernel");
                cudaCheckError(hipDeviceSynchronize(),"kernel execution failed");
                cudaCheckError(hipMemcpy(&host_result,device_result,sizeof(int),hipMemcpyDeviceToHost),"failed to copy data from device memory to host memory");
                cudaCheckError(hipFree(device_result),"failed to free the memory from the device memory");
                return host_result;
            }

            int ne(const Var &other) const {
                int host_result;
                int *device_result;
                cudaCheckError(hipMalloc(&device_result,sizeof(int)),"failed to allocate the memory on GPU");
                neKernel<<<1,1>>>(this -> value,other.value,device_result);
                cudaCheckError(hipGetLastError(),"failed to launch neKernel");
                cudaCheckError(hipDeviceSynchronize(),"kernel execution failed");
                cudaCheckError(hipMemcpy(&host_result,device_result,sizeof(int),hipMemcpyDeviceToHost),"failed to copy data from device memory to host memory");
                cudaCheckError(hipFree(device_result),"failed to free the memory from the device memory");
                return host_result;
            }

            int gt(const Var &other) const {
                int host_result;
                int *device_result;
                cudaCheckError(hipMalloc(&device_result,sizeof(int)),"failed to allocate the memory on GPU");
                gtKernel<<<1,1>>>(this -> value,other.value,device_result);
                cudaCheckError(hipGetLastError(),"failed to launch gtKernel");
                cudaCheckError(hipDeviceSynchronize(),"kernel execution failed");
                cudaCheckError(hipMemcpy(&host_result,device_result,sizeof(int),hipMemcpyDeviceToHost),"failed to copy data from device memory to host memory");
                cudaCheckError(hipFree(device_result),"failed to free the memory from the device memory");
                return host_result;
            }

            int ge(const Var &other) const {
                int host_result;
                int *device_result;
                cudaCheckError(hipMalloc(&device_result,sizeof(int)),"failed to allocate the memory on GPU");
                geKernel<<<1,1>>>(this -> value,other.value,device_result);
                cudaCheckError(hipGetLastError(),"failed to lauch geKernel");
                cudaCheckError(hipDeviceSynchronize(),"kenrel execution failed");
                cudaCheckError(hipMemcpy(&host_result,device_result,sizeof(int),hipMemcpyDeviceToHost),"falied to copy data from device memory to host memry");
                cudaCheckError(hipFree(device_result),"failed to free the memory from the device memory");
                return host_result;
            }

            int lt(const Var &other) const {
                int host_result;
                int *device_result;
                cudaCheckError(hipMalloc(&device_result,sizeof(int)),"failed to allocate the memory on GPU");
                ltKernel<<<1,1>>>(this -> value,other.value,device_result);
                cudaCheckError(hipGetLastError(),"failed to launch ltKernel");
                cudaCheckError(hipDeviceSynchronize(),"kernel execution failed");
                cudaCheckError(hipMemcpy(&host_result,device_result,sizeof(int),hipMemcpyDeviceToHost),"failed to copy data from device memory to host memory");
                cudaCheckError(hipFree(device_result),"failed to free the memory from the device memory");
                return host_result;
            }

            int le(const Var &other) const {
                int host_result;
                int *device_result;
                cudaCheckError(hipMalloc(&device_result,sizeof(int)),"failed to allocate the memory on GPU");
                leKernel<<<1,1>>>(this -> value,other.value,device_result);
                cudaCheckError(hipGetLastError(),"failed to lauch leKernel");
                cudaCheckError(hipDeviceSynchronize(),"kenrel execution failed");
                cudaCheckError(hipMemcpy(&host_result,device_result,sizeof(int),hipMemcpyDeviceToHost),"falied to copy data from device memory to host memry");
                cudaCheckError(hipFree(device_result),"failed to free the memory from the device memory");
                return host_result;
            }

            Var abs() const {
                Var result(0.0);
                absKernel<<<1,1>>>(this -> value,result.value);
                cudaCheckError(hipGetLastError(),"failed to launch absKernel");
                cudaCheckError(hipDeviceSynchronize(),"kernel execution failed");
                return result;
            }

            Var sqrt() const {
                Var result(0.0);
                sqrtKernel<<<1,1>>>(this -> value,result.value);
                cudaCheckError(hipGetLastError(),"failed to launch sqrtKernel");
                cudaCheckError(hipDeviceSynchronize(),"kernel execution failed");
                return result;
            }

            Var cbrt() const {
                Var result(0.0);
                cbrtKernel<<<1,1>>>(this -> value,result.value);
                cudaCheckError(hipGetLastError(),"failed to launch cbrtKernel");
                cudaCheckError(hipDeviceSynchronize(),"kernel execution failed");
                return result;
            }

            Var fact() const {
                Var result(0.0);
                factKernel<<<1,1>>>(this -> value,result.value);
                cudaCheckError(hipGetLastError(),"failed to launch factKerel");
                cudaCheckError(hipDeviceSynchronize(),"kernel execution failed");
                return result;
            }

            Var neg() const {
                Var result(0.0);
                negKernel<<<1,1>>>(this -> value,result.value);
                cudaCheckError(hipGetLastError(),"failed to launch negKernel");
                cudaCheckError(hipDeviceSynchronize(),"kernel execution failed");
                return result;
            }

            Var pos() const {
                Var result(0.0);
                posKernel<<<1,1>>>(this -> value,result.value);
                cudaCheckError(hipGetLastError(),"failed to launch posKernel");
                cudaCheckError(hipDeviceSynchronize(),"kernel execution failed");
                return result;
            }
    };

    Var *Var_new(double data){ return new Var(data); }
    void Var_delete(Var* var){ delete var; }
    double Var_get_value(Var* var){ return var -> get_value(); }

    Var *Var_add(Var *a,Var *b){ return new Var(a -> add(*b)); }
    Var *Var_sub(Var *a,Var *b){ return new Var(a -> sub(*b)); }
    Var *Var_mul(Var *a,Var *b){ return new Var(a -> mul(*b)); }
    Var *Var_tdiv(Var *a,Var *b){ return new Var(a -> tdiv(*b)); }
    Var *Var_fdiv(Var *a,Var *b){ return new Var(a -> fdiv(*b)); }
    Var *Var_pow(Var *a,Var *b){ return new Var(a -> pow(*b)); }
    Var *Var_mod(Var *a,Var *b){ return new Var(a -> mod(*b)); }

    int Var_eq(Var *a,Var *b){ return a -> eq(*b); }
    int Var_ne(Var *a,Var *b){ return a -> ne(*b); }
    int Var_gt(Var *a,Var *b){ return a -> gt(*b); }
    int Var_ge(Var *a,Var *b){ return a -> ge(*b); }
    int Var_lt(Var *a,Var *b){ return a -> lt(*b); }
    int Var_le(Var *a,Var *b){ return a -> le(*b); }

    Var *Var_abs(Var *a){ return new Var(a -> abs()); }
    Var *Var_sqrt(Var *a){ return new Var(a -> sqrt()); }
    Var *Var_cbrt(Var *a){ return new Var(a -> cbrt()); }
    Var *Var_fact(Var *a){ return new Var(a -> fact()); }
    Var *Var_neg(Var *a){ return new Var(a -> neg()); }
    Var *Var_pos(Var *a){ return new Var(a -> pos()); }
}


